#include "hip/hip_runtime.h"
//
// Created by Sahar on 22/07/2022.
//

#include "LayerHUD.cuh"
#include "utils.h"

#include <imgui-docking/include/imgui.h>
#include <imgui-docking/include/imgui_impl_glfw.h>
#include <imgui-docking/include/imgui_impl_opengl3.h>


LayerHUD::LayerHUD(GLFWwindow *window,
                   std::shared_ptr<LayerRGBStream> layerRGB,
                   std::string glslVersion) : _window(window), _layerRGB(std::move(layerRGB)),
                                              _glsl_version(std::move(glslVersion)) {
}

void LayerHUD::onUpdate() {
    ImGui::Begin("HUD Information", nullptr, ImGuiWindowFlags_::ImGuiWindowFlags_AlwaysAutoResize);

    imGuiFpsInfo();
    imGuiCameraSettings();
    imGuiRayTracerSettings();
    imGuiSaveImageSection();

    ImGui::End();
}

void LayerHUD::imGuiRayTracerSettings() {
    if (ImGui::CollapsingHeader("Ray Tracer settings:")) {
        imGuiNRenderCallsSlider();
        imGuiRenderWidthButtons();
        imGuiNRayBouncesSlider();
    }
}

void LayerHUD::imGuiNRayBouncesSlider() const {
    static int nRayBounces = _layerRGB->getRayTracerRenderer()->getNRayBounces();
    ImGui::SliderInt("# Ray Bounces", &nRayBounces, 1, 12);
    if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled))
    {
        ImGui::SetTooltip("Number of RayTracer's Bounces for each ray cast into the scene..");
    }
    if (nRayBounces != _layerRGB->getRayTracerRenderer()->getNRayBounces()) {
        _layerRGB->getRayTracerRenderer()->setNRayBounces(nRayBounces);
    }
}

void LayerHUD::imGuiRenderWidthButtons() {
    ImGui::Text("Render width:");
    static int renderWidth = _layerRGB->getRendererImageWidth();
    imGuiRenderWidthButton(renderWidth, "80", 80);
    imGuiSameLineSpace();
    imGuiRenderWidthButton(renderWidth, "240", 240);
    imGuiSameLineSpace();
    imGuiRenderWidthButton(renderWidth, "400", 400);
    imGuiSameLineSpace();
    imGuiRenderWidthButton(renderWidth, "800", 800);

    imGuiRenderWidthButton(renderWidth, "1200", 1200);
    imGuiSameLineSpace();
    imGuiRenderWidthButton(renderWidth, "1600", 1600);
    imGuiSameLineSpace();
    imGuiRenderWidthButton(renderWidth, "2000", 2000);

    if (renderWidth != _layerRGB->getRendererImageWidth()) {
        _layerRGB->setRendererImageWidth(renderWidth);
    }
}

void LayerHUD::imGuiNRenderCallsSlider() {
    _nRenders = _layerRGB->getRendersPerFrame();
    ImGui::SliderInt("# RayTracer Render Calls per Frame", &_nRenders, 1, 15);
    if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled))
    {
        ImGui::SetTooltip("Number of RayTracer Render Calls that will be made for each loop iteration in the main application GUI loop.");
    }
    if (_nRenders != _layerRGB->getRendersPerFrame()) {
        _layerRGB->setRendersPerFrame(_nRenders);
    }
}

void LayerHUD::imGuiRenderWidthButton(int & renderWidth, const char *buttonLabel, int afterPressWidth) const {
    if (ImGui::Button(buttonLabel)) {
        renderWidth = afterPressWidth;
    }
    if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled)) {
        ImGui::SetTooltip("Width of Image that the ray tracer will render.");
    }
}

void LayerHUD::imGuiSameLineSpace() const {
    ImGui::SameLine(0.0f, ImGui::GetStyle().ItemInnerSpacing.x);
}

void LayerHUD::imGuiCameraSettings() {
    if (ImGui::CollapsingHeader("Camera settings:", ImGuiTreeNodeFlags_FramePadding)) {
        imGuiCameraVFovSlider();
        imGuiCameraFocusDistSlider();
        imGuiCameraApertureSlider();
    }
}

void LayerHUD::imGuiCameraVFovSlider() {
    static float cameraVFov = _layerRGB->getCameraVFov();
    ImGui::SliderFloat("Camera Vertical Fov", &cameraVFov, 1.0f, 170.f);
    if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled)) {
        ImGui::SetTooltip("Camera Vertical Fov measured in degrees.");
    }
    if (fabs(cameraVFov - _layerRGB->getCameraVFov()) > 0.00001f) {
        _layerRGB->setCameraVFov(cameraVFov);
    }
}

void LayerHUD::imGuiCameraFocusDistSlider() {
    static float cameraFocusDist = _layerRGB->getCameraFocusDist();
    ImGui::SliderFloat("Camera Focus Distance", &cameraFocusDist, 0.01f, 50.f);
    if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled)) {
        ImGui::SetTooltip("Distance from the \"eye\" of the camera towards the scene.");
    }
    if (fabs(cameraFocusDist - _layerRGB->getCameraFocusDist()) > 0.00001f) {
        _layerRGB->setCameraFocusDist(cameraFocusDist);
    }
}

void LayerHUD::imGuiCameraApertureSlider() {
    static float cameraAperture = _layerRGB->getCameraAperture();
    ImGui::SliderFloat("Camera Aperture", &cameraAperture, 0.00001f, 0.1f);
    if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled)) {
        ImGui::SetTooltip("The smaller the aperture, the sharper the image.");
    }
    if (fabs(cameraAperture - _layerRGB->getCameraAperture()) > 0.00001f) {
        _layerRGB->setCameraAperture(cameraAperture);
    }
}

void LayerHUD::imGuiFpsInfo() {
    ImGui::Text("Application - average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate,
                ImGui::GetIO().Framerate);
    _nRenders = _layerRGB->getRendersPerFrame();
    ImGui::Text("RayTracer   - average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate / _nRenders,
                ImGui::GetIO().Framerate * _nRenders);
}

void LayerHUD::imGuiSaveImageSection() const {
    if (ImGui::CollapsingHeader("Save:")) {
        static const int bufSize = 64;
        static char filename[64];
        static bool isFirstInit = false;
        if (isFirstInit) {
            strncpy(filename, "test.jpg", bufSize);
            isFirstInit = true;
        }
        ImGui::InputText("##", filename, bufSize);
        if (ImGui::IsItemHovered(ImGuiHoveredFlags_AllowWhenDisabled)) {
            ImGui::SetTooltip("Name of the file to save. Will be saved in the root directory of the executable.");
        }

        imGuiSameLineSpace();

        if (ImGui::Button("Save Image")) {
            saveImage(filename);
        }
    }
}

void LayerHUD::saveImage(const std::string &filename) const {
    auto rayTracer = _layerRGB->getRayTracerRenderer();
    saveImgAsJpg(filename, rayTracer->getPixelsOutAsChars(), rayTracer->getImgW(), rayTracer->getImgH());
}


void LayerHUD::onAttach() {
    ImGui::CreateContext();
    ImGui_ImplGlfw_InitForOpenGL(getWindow(), true);
    ImGui_ImplOpenGL3_Init(_glsl_version.c_str());
    ImGui::StyleColorsDark();
}

void LayerHUD::onDetach() {
}

GLFWwindow *LayerHUD::getWindow() const {
    return _window;
}

