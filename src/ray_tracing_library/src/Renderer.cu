#include "hip/hip_runtime.h"
//
// Created by Sahar on 10/06/2022.
//

#include <vector>
#include <Renderer.cuh>
#include "utils.cuh"
#include <iostream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
//         Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


__global__
void createWorld(World **deviceWorld, Sphere *spheres, size_t nSpheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *deviceWorld = new World(spheres, nSpheres);
    }
}


__global__
void freeWorld(World **deviceWorld) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *deviceWorld;
    }
}

__global__
void getPixel(Color *pixel,
              Camera c,
              World **d_world,
              int nSamplesPerPixel,
              int row,
              int col,
              int *randState,
              int imWidth,
              int imHeight,
              int nBounces) {
    Color pixelSum{0,0,0};
    for (int i = 0; i < nSamplesPerPixel; ++i) {
        auto h = (static_cast<double>(col) + randomDouble(*randState)) / (imWidth - 1);
        auto v = 1 - ((static_cast<double>(row) + randomDouble(*randState)) / (imHeight - 1));
        Ray ray = c.getRay(h, v, *randState);
        pixelSum += (*d_world)->rayTrace(ray, nBounces, *randState);
    }
    *pixel = pixelSum;
}

void getPixel2(Color &pixel,
               const Camera &c,
               const World &d_world,
               int nSamplesPerPixel,
               int row,
               int col,
               int *randState,
               int imWidth,
               int imHeight,
               int nBounces) {
    Color pixelSum{0,0,0};
    for (int i = 0; i < nSamplesPerPixel; ++i) {
        auto h = (static_cast<double>(col) + randomDouble(*randState)) / (imWidth - 1);
        auto v = 1 - ((static_cast<double>(row) + randomDouble(*randState)) / (imHeight - 1));
        Ray ray = c.getRay(h, v, *randState);
        pixelSum += d_world.rayTrace(ray, nBounces, *randState);
    }
    pixel = pixelSum / nSamplesPerPixel;
}

std::vector<Color> Renderer::render() const {
    World **d_world = allocateWorldInDeviceMemory(_world.getSpheres(), _world.getNSpheres());

    std::vector<Color> data(_imageHeight * _imageWidth, {1, 1, 1});
    for (int pixel_idx = 0; pixel_idx < _imageHeight * _imageWidth; ++pixel_idx) {
        int row = pixel_idx / _imageWidth;
        int col = pixel_idx % _imageWidth;

        Color *pixelRes;
        int *randState;
        checkCudaErrors(hipMallocManaged(&pixelRes, sizeof(Color) * 1));
        checkCudaErrors(hipMallocManaged(&randState, sizeof(int) * 1));


        getPixel<<<1, 1>>>(pixelRes, _camera, d_world, _nSamplesPerPixel, row, col, randState, _imageWidth, _imageHeight, _nRayBounces);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        Vec3 pixelColor = *pixelRes;
        checkCudaErrors(hipFree(pixelRes));
        checkCudaErrors(hipFree(randState));

//        // TODO-Sahar: Remove:
//        Vec3 pixelColor{};
//        int demoRand = 0;
//        getPixel2(pixelColor, _camera, _world, _nSamplesPerPixel, row, col, &demoRand, _imageWidth, _imageHeight, _nRayBounces);

        pixelColor = clamp(gammaCorrection(pixelColor), 0.0, 0.999);
        data[row * _imageWidth + col] = pixelColor;
    }
    freeWorldFromDeviceAndItsPtr(d_world);
    return data;
}

void Renderer::freeWorldFromDeviceAndItsPtr(World **d_world) {
    // Free world object from the device.
    freeWorld<<<1,1>>>(d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Free the pointer to the world object.
    checkCudaErrors(hipFree(d_world));
}

World **Renderer::allocateWorldInDeviceMemory(const Sphere *ptrSpheres, size_t nSpheres) {
    // Copy the sphereArr to the GPU memory.
    Sphere * sphereArr;
    checkCudaErrors(hipMallocManaged((void **) &sphereArr, sizeof(Sphere) * nSpheres));
    for (int i = 0; i < nSpheres; ++i) {
        sphereArr[i] = ptrSpheres[i];
    }

    // Create d_world with the sphereArr.
    World **d_world;
    checkCudaErrors(hipMallocManaged(&d_world, sizeof(World *) * 1));
    createWorld<<<1,1>>>(d_world, sphereArr, nSpheres);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Free the sphereArr memory from the GPU memory.
    checkCudaErrors(hipFree(sphereArr));
    return d_world;
}
