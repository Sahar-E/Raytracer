#include "hip/hip_runtime.h"
//
// Created by Sahar on 10/06/2022.
//

#include <vector>
#include <Renderer.cuh>
#include "utils.cuh"
#include <iostream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )


void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
//         Make sure we call CUDA Device Reset before exiting
        std::cerr << "CUDA hipGetErrorString: " << hipGetErrorString(result) << "\n";
        hipDeviceReset();
        exit(99);
    }
}

void freeWorldFromDeviceAndItsPtr2(World **d_world);
World **allocateWorldInDeviceMemory2(const Sphere *ptrSpheres, size_t nSpheres);



__global__
void createWorld(World **deviceWorld, Sphere *spheres, size_t nSpheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *deviceWorld = new World(spheres, nSpheres);
    }
}


__global__
void freeWorld(World **deviceWorld) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *deviceWorld;
    }
}

__global__
void getPixel(Color *pixel,
              Camera c,
              World **d_world,
              int nSamplesPerPixel,
              int row,
              int col,
              int *randState,
              int imWidth,
              int imHeight,
              int nBounces) {
    Color pixelSum{0, 0, 0};
    for (int i = 0; i < nSamplesPerPixel; ++i) {
        auto h = (static_cast<double>(col) + randomDouble(*randState)) / (imWidth - 1);
        auto v = 1 - ((static_cast<double>(row) + randomDouble(*randState)) / (imHeight - 1));
        Ray ray = c.getRay(h, v, *randState);
        pixelSum += (*d_world)->rayTrace(ray, nBounces, *randState);
    }
    *pixel = pixelSum;
}

void getPixel2(Color &pixel,
               const Camera &c,
               const World &d_world,
               int nSamplesPerPixel,
               int row,
               int col,
               int *randState,
               int imWidth,
               int imHeight,
               int nBounces) {
    Color pixelSum{0, 0, 0};
    for (int i = 0; i < nSamplesPerPixel; ++i) {
        auto h = (static_cast<double>(col) + randomDouble(*randState)) / (imWidth - 1);
        auto v = 1 - ((static_cast<double>(row) + randomDouble(*randState)) / (imHeight - 1));
        Ray ray = c.getRay(h, v, *randState);
        pixelSum += d_world.rayTrace(ray, nBounces, *randState);
    }
    pixel = pixelSum / nSamplesPerPixel;
}


__global__
void writePixels(Color *pixelsOut,
                 Camera c,
                 World **d_world,
                 int nSamplesPerPixel,
                 int *randStates,
                 int imWidth,
                 int imHeight,
                 int nBounces) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int pixel_idx = index; pixel_idx < imHeight * imWidth; pixel_idx += stride) {
        int row = pixel_idx / imWidth;
        int col = pixel_idx % imWidth;
        if (threadIdx.x == 0) {
            printf("Rows left to process: %d\n\r", imHeight - row);
        }
        Color pixelSum{0, 0, 0};
        for (int i = 0; i < nSamplesPerPixel; ++i) {
            auto h = (static_cast<double>(col) + randomDouble(randStates[pixel_idx])) / (imWidth - 1);
            auto v = 1 - ((static_cast<double>(row) + randomDouble(randStates[pixel_idx])) / (imHeight - 1));
            Ray ray = c.getRay(h, v, randStates[pixel_idx]);
            pixelSum += (*d_world)->rayTrace(ray, nBounces, randStates[pixel_idx]);
        }
        Color pixelAverage = pixelSum / nSamplesPerPixel;
        pixelsOut[pixel_idx] = clamp(gammaCorrection(pixelAverage), 0.0, 0.999);
    }
}

void writePixels2(Color *pixelsOut,
                 Camera c,
                 World **d_world,
                 int nSamplesPerPixel,
                 int *randStates,
                 int imWidth,
                 int imHeight,
                 int nBounces) {
    for (int pixel_idx = 1; pixel_idx < imHeight * imWidth; pixel_idx += 1) {
        int row = pixel_idx / imWidth;
        int col = pixel_idx % imWidth;
        printf("Rows left to process: %d\n\r", imHeight - row);
        Color pixelSum{0, 0, 0};
        for (int i = 0; i < nSamplesPerPixel; ++i) {
            auto h = (static_cast<double>(col) + randomDouble(randStates[pixel_idx])) / (imWidth - 1);
            auto v = 1 - ((static_cast<double>(row) + randomDouble(randStates[pixel_idx])) / (imHeight - 1));
            Ray ray = c.getRay(h, v, randStates[pixel_idx]);
            pixelSum += (*d_world)->rayTrace(ray, nBounces, randStates[pixel_idx]);
        }
        Color pixelAverage = pixelSum / nSamplesPerPixel;
        pixelsOut[pixel_idx] = clamp(gammaCorrection(pixelAverage), 0.0, 0.999);
    }
}

std::vector<Color> Renderer::render() const {
    Color *pixelsOut;
    int *randStates;
    int nPixels = _imageHeight * _imageWidth;
    checkCudaErrors(hipMallocManaged(&pixelsOut, sizeof(Color) * nPixels));
    checkCudaErrors(hipMallocManaged(&randStates, sizeof(int) * nPixels));
    World **d_world = allocateWorldInDeviceMemory(_world.getSpheres(), _world.getNSpheres());


//    pixelsOut = new Color[nPixels];
//    randStates = new int[nPixels]();

//    World **d_world = allocateWorldInDeviceMemory2(_world.getSpheres(), _world.getNSpheres());

    int blockSize = 512;
    int numBlocks = (nPixels + blockSize - 1) / blockSize;
    writePixels<<<numBlocks, blockSize>>>(pixelsOut, _camera, d_world, _nSamplesPerPixel, randStates, _imageWidth, _imageHeight,_nRayBounces);

//    writePixels2(pixelsOut, _camera, d_world, _nSamplesPerPixel, randStates, _imageWidth, _imageHeight,_nRayBounces);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    freeWorldFromDeviceAndItsPtr(d_world);
//    freeWorldFromDeviceAndItsPtr2(d_world);

    std::vector<Color> data(nPixels, {0, 1, 0});
    // copy the pixels into data:
    for (int i = 0; i < nPixels; i++) {
        data[i] = pixelsOut[i];
    }

    checkCudaErrors(hipFree(pixelsOut));
    checkCudaErrors(hipFree(randStates));

//    delete[] pixelsOut;
//    delete[] randStates;


//    for (int pixel_idx = 0; pixel_idx < _imageHeight * _imageWidth; ++pixel_idx) {
//        int row = pixel_idx / _imageWidth;
//        int col = pixel_idx % _imageWidth;

//        Color *pixelRes;
//        int *randState;
//        checkCudaErrors(hipMallocManaged(&pixelRes, sizeof(Color) * 1));
//        checkCudaErrors(hipMallocManaged(&randState, sizeof(int) * 1));


//        getPixel<<<1, 1>>>(pixelRes, _camera, d_world, _nSamplesPerPixel, row, col, randState, _imageWidth, _imageHeight, _nRayBounces);
//        checkCudaErrors(hipGetLastError());
//        checkCudaErrors(hipDeviceSynchronize());

//        Vec3 pixelColor = *pixelRes;        // Copy the color to the stack.
//        checkCudaErrors(hipFree(pixelRes));
//        checkCudaErrors(hipFree(randState));

//        // TODO-Sahar: Remove:
//        Vec3 pixelColor{};
//        int demoRand = 0;
//        getPixel2(pixelColor, _camera, _world, _nSamplesPerPixel, row, col, &demoRand, _imageWidth, _imageHeight, _nRayBounces);

//        pixelColor = clamp(gammaCorrection(pixelColor), 0.0, 0.999);
//        data[row * _imageWidth + col] = pixelColor;
//    }


    return data;
}




std::vector<Color> Renderer::render2() const {
    Color *pixelsOut;
    int *randStates;
    int nPixels = _imageHeight * _imageWidth;
    pixelsOut = new Color[nPixels];
    randStates = new int[nPixels]();

    World **d_world = allocateWorldInDeviceMemory2(_world.getSpheres(), _world.getNSpheres());
    writePixels2(pixelsOut, _camera, d_world, _nSamplesPerPixel, randStates, _imageWidth, _imageHeight,_nRayBounces);

    freeWorldFromDeviceAndItsPtr2(d_world);

    std::vector<Color> data(nPixels, {0, 1, 0});
    // copy the pixels into data:
    for (int i = 0; i < nPixels; i++) {
        data[i] = pixelsOut[i];
    }


    delete[] pixelsOut;
    delete[] randStates;

return data;
}

void Renderer::freeWorldFromDeviceAndItsPtr(World **d_world) {
    // Free world object from the device.
    freeWorld<<<1, 1>>>(d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Free the pointer to the world object.
    checkCudaErrors(hipFree(d_world));
}

void freeWorldFromDeviceAndItsPtr2(World **d_world) {
    // Free world object from the device.
    delete *d_world;
    // Free the pointer to the world object.
    delete d_world;
}

World **Renderer::allocateWorldInDeviceMemory(const Sphere *ptrSpheres, size_t nSpheres) {
    // Copy the sphereArr to the GPU memory.
    Sphere *sphereArr;
    checkCudaErrors(hipMallocManaged((void **) &sphereArr, sizeof(Sphere) * nSpheres));
    for (int i = 0; i < nSpheres; ++i) {
        sphereArr[i] = ptrSpheres[i];
    }

    // Create d_world with the sphereArr.
    World **d_world;
    checkCudaErrors(hipMallocManaged(&d_world, sizeof(World *) * 1));
    createWorld<<<1, 1>>>(d_world, sphereArr, nSpheres);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Free the sphereArr memory from the GPU memory.
    checkCudaErrors(hipFree(sphereArr));
    return d_world;
}

World **allocateWorldInDeviceMemory2(const Sphere *ptrSpheres, size_t nSpheres) {
    auto **d_world = new World*;
    *d_world = new World(ptrSpheres, nSpheres);
    return d_world;
}
