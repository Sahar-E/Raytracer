#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <World.cuh>
#include <Camera.cuh>
#include <Renderer.cuh>
#include "utils.cuh"
#include "TimeThis.h"
#include "commonDefines.h"
#include <string>
#include <cassert>
#include "hip/hip_runtime_api.h"
#include "commonCuda.cuh"


int main() {
    const auto aspectRatio = 3.0f / 2.0f;
    const int image_width = 1200;
    const int image_height = static_cast<int>(image_width / aspectRatio);
    const int rayBounces = 7;
    float vFov = 26.0f;
    float aperture = 0.05f;
    float focusDist = 10.0f;
    int nFrames = 1000;

    assert(0 < rayBounces && rayBounces <= MAX_BOUNCES);

    Vec3 vUp = {0, 1, 0};
    Vec3 lookFrom = {0, 1.8, 12};
    Vec3 lookAt = {0., 0, 0};


    auto world = World::initWorld2();
    std::cout << "Size: " << world.getTotalSizeInMemoryForObjects() << "\n";
    std::cout << "nSpheres: " << world.getNSpheres()  << "\n";
    assert(world.getTotalSizeInMemoryForObjects() < 48 * pow(2, 10) && "There is a hard limit for NVIDIA's shared memory size of 48KB for one block.");
    auto camera = Camera(lookFrom, lookAt, vUp, aspectRatio, vFov, aperture, focusDist);
    Renderer renderer(image_width, image_height, world, camera, rayBounces);

    for (int j = 0; j < nFrames; ++j) {
        renderer.render();
        std::cout << "Done iteration #: " << j  << "\n";
    }

    std::string filename = "test.jpg";
    int channelCount = 3;
    std::vector<std::tuple<float, float, float>> rgb(renderer.getNPixelsOut(), {0, 0, 0});
    for (int i = 0; i < renderer.getNPixelsOut(); ++i) {
        Color pixel = renderer.getPixelsOut()[i];
        pixel = clamp(gammaCorrection(pixel), 0.0, 0.999);
        rgb[i] = {pixel.x(), pixel.y(), pixel.z()};
    }
    saveImgAsJpg(filename, rgb, image_width, image_height, channelCount);

    std::cout << "Done." << "\n";
    return 0;
}